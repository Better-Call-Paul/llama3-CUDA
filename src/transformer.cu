#include "transformer.cuh"
#include <cstdio>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include "rms_norm.cuh"
#include "rope_rotation.cuh"
#include "mat_mul.cuh"
#include "multi_head_attention.cuh"

namespace llama {

#define CUDA_CHECK(val) { \
    if (val != hipSuccess) { \
        throw std::runtime_error(std::string("CUDA Error: ") + hipGetErrorString(val)); \
    } \
}


}